#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define ind2d(i, j) (i) * (tam + 2) + j
#define POWMIN 3
#define POWMAX 10

// Macro para verificacao de erros CUDA
#define CUDA_CHECK(call)                                             \
    do                                                               \
    {                                                                \
        hipError_t err = call;                                      \
        if (err != hipSuccess)                                      \
        {                                                            \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                   hipGetErrorString(err));                         \
            exit(EXIT_FAILURE);                                      \
        }                                                            \
    } while (0)

double wall_time(void)
{
    struct timeval tv;
    struct timezone tz;

    gettimeofday(&tv, &tz);
    return (tv.tv_sec + tv.tv_usec / 1000000.0);
}

/* Kernel CUDA para aplicar as regras do Jogo da Vida */
__global__ void UmaVidaCUDA(int *tabulIn, int *tabulOut, int tam)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1; // +1 para pular a borda
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1; // +1 para pular a borda

    // Verificar se estamos dentro dos limites validos
    if (i <= tam && j <= tam)
    {
        int vizviv = tabulIn[ind2d(i - 1, j - 1)] + tabulIn[ind2d(i - 1, j)] +
                     tabulIn[ind2d(i - 1, j + 1)] + tabulIn[ind2d(i, j - 1)] +
                     tabulIn[ind2d(i, j + 1)] + tabulIn[ind2d(i + 1, j - 1)] +
                     tabulIn[ind2d(i + 1, j)] + tabulIn[ind2d(i + 1, j + 1)];

        if (tabulIn[ind2d(i, j)] && vizviv < 2)
            tabulOut[ind2d(i, j)] = 0;
        else if (tabulIn[ind2d(i, j)] && vizviv > 3)
            tabulOut[ind2d(i, j)] = 0;
        else if (!tabulIn[ind2d(i, j)] && vizviv == 3)
            tabulOut[ind2d(i, j)] = 1;
        else
            tabulOut[ind2d(i, j)] = tabulIn[ind2d(i, j)];
    }
}

/* Kernel otimizado com memoria compartilhada */
__global__ void UmaVidaCUDA_Shared(int *tabulIn, int *tabulOut, int tam)
{
    // Definir memoria compartilhada com halo
    __shared__ int shared_tabul[18][18]; // 16x16 + 2 de halo em cada direcao

    int global_i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int global_j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int local_i = threadIdx.y + 1;
    int local_j = threadIdx.x + 1;

    // Carregar dados para memoria compartilhada
    if (global_i <= tam + 1 && global_j <= tam + 1)
    {
        shared_tabul[local_i][local_j] = tabulIn[ind2d(global_i, global_j)];

        // Carregar bordas
        if (threadIdx.y == 0 && global_i > 0)
        {
            shared_tabul[0][local_j] = tabulIn[ind2d(global_i - 1, global_j)];
        }
        if (threadIdx.y == blockDim.y - 1 && global_i < tam + 1)
        {
            shared_tabul[local_i + 1][local_j] = tabulIn[ind2d(global_i + 1, global_j)];
        }
        if (threadIdx.x == 0 && global_j > 0)
        {
            shared_tabul[local_i][0] = tabulIn[ind2d(global_i, global_j - 1)];
        }
        if (threadIdx.x == blockDim.x - 1 && global_j < tam + 1)
        {
            shared_tabul[local_i][local_j + 1] = tabulIn[ind2d(global_i, global_j + 1)];
        }
    }

    __syncthreads();

    // Processar apenas celulas validas
    if (global_i <= tam && global_j <= tam)
    {
        int vizviv = shared_tabul[local_i - 1][local_j - 1] + shared_tabul[local_i - 1][local_j] +
                     shared_tabul[local_i - 1][local_j + 1] + shared_tabul[local_i][local_j - 1] +
                     shared_tabul[local_i][local_j + 1] + shared_tabul[local_i + 1][local_j - 1] +
                     shared_tabul[local_i + 1][local_j] + shared_tabul[local_i + 1][local_j + 1];

        int current = shared_tabul[local_i][local_j];

        if (current && vizviv < 2)
            tabulOut[ind2d(global_i, global_j)] = 0;
        else if (current && vizviv > 3)
            tabulOut[ind2d(global_i, global_j)] = 0;
        else if (!current && vizviv == 3)
            tabulOut[ind2d(global_i, global_j)] = 1;
        else
            tabulOut[ind2d(global_i, global_j)] = current;
    }
}

void DumpTabul(int *tabul, int tam, int first, int last, char *msg)
{
    int i, ij;

    printf("%s; Dump posicoes [%d:%d, %d:%d] de tabuleiro %d x %d\n",
           msg, first, last, first, last, tam, tam);
    for (i = first; i <= last; i++)
        printf("=");
    printf("=\n");
    for (i = ind2d(first, 0); i <= ind2d(last, 0); i += ind2d(1, 0))
    {
        for (ij = i + first; ij <= i + last; ij++)
            printf("%c", tabul[ij] ? 'X' : '.');
        printf("\n");
    }
    for (i = first; i <= last; i++)
        printf("=");
    printf("=\n");
}

void InitTabul(int *tabulIn, int *tabulOut, int tam)
{
    int ij;

    for (ij = 0; ij < (tam + 2) * (tam + 2); ij++)
    {
        tabulIn[ij] = 0;
        tabulOut[ij] = 0;
    }

    tabulIn[ind2d(1, 2)] = 1;
    tabulIn[ind2d(2, 3)] = 1;
    tabulIn[ind2d(3, 1)] = 1;
    tabulIn[ind2d(3, 2)] = 1;
    tabulIn[ind2d(3, 3)] = 1;
}

int Correto(int *tabul, int tam)
{
    int ij, cnt;

    cnt = 0;
    for (ij = 0; ij < (tam + 2) * (tam + 2); ij++)
        cnt = cnt + tabul[ij];
    return (cnt == 5 && tabul[ind2d(tam - 2, tam - 1)] &&
            tabul[ind2d(tam - 1, tam)] && tabul[ind2d(tam, tam - 2)] &&
            tabul[ind2d(tam, tam - 1)] && tabul[ind2d(tam, tam)]);
}

int main(void)
{
    int pow;
    int i, tam, *h_tabulIn, *h_tabulOut; // Host arrays
    int *d_tabulIn, *d_tabulOut;         // Device arrays
    double t0, t1, t2, t3;

    // Informacoes da GPU
    int deviceCount;
    CUDA_CHECK(hipGetDeviceCount(&deviceCount));

    if (deviceCount == 0)
    {
        printf("Nenhuma GPU CUDA encontrada!\n");
        return -1;
    }

    hipDeviceProp_t deviceProp;
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("Usando GPU: %s\n", deviceProp.name);
    printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Memoria Global: %lu bytes\n", deviceProp.totalGlobalMem);

    // Loop para todos os tamanhos do tabuleiro
    for (pow = POWMIN; pow <= POWMAX; pow++)
    {
        tam = 1 << pow;
        int total_size = (tam + 2) * (tam + 2) * sizeof(int);

        printf("\n--- Processando tabuleiro %dx%d ---\n", tam, tam);

        // Alocacao na CPU
        t0 = wall_time();
        h_tabulIn = (int *)malloc(total_size);
        h_tabulOut = (int *)malloc(total_size);

        // Alocacao na GPU
        CUDA_CHECK(hipMalloc((void **)&d_tabulIn, total_size));
        CUDA_CHECK(hipMalloc((void **)&d_tabulOut, total_size));

        // Inicializacao
        InitTabul(h_tabulIn, h_tabulOut, tam);

        // Copia inicial para GPU
        CUDA_CHECK(hipMemcpy(d_tabulIn, h_tabulIn, total_size, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_tabulOut, h_tabulOut, total_size, hipMemcpyHostToDevice));

        t1 = wall_time();

        // Configuracao dos blocos e threads
        dim3 blockSize(16, 16);
        dim3 gridSize((tam + blockSize.x - 1) / blockSize.x,
                      (tam + blockSize.y - 1) / blockSize.y);

        printf("Grid: %dx%d, Block: %dx%d\n", gridSize.x, gridSize.y, blockSize.x, blockSize.y);

        // Loop principal de evolucao
        for (i = 0; i < 2 * (tam - 3); i++)
        {
            // Primeira evolucao: d_tabulIn -> d_tabulOut
            UmaVidaCUDA_Shared<<<gridSize, blockSize>>>(d_tabulIn, d_tabulOut, tam);
            CUDA_CHECK(hipDeviceSynchronize());

            // Segunda evolucao: d_tabulOut -> d_tabulIn
            UmaVidaCUDA_Shared<<<gridSize, blockSize>>>(d_tabulOut, d_tabulIn, tam);
            CUDA_CHECK(hipDeviceSynchronize());
        }

        // Copia resultado de volta para CPU
        CUDA_CHECK(hipMemcpy(h_tabulIn, d_tabulIn, total_size, hipMemcpyDeviceToHost));

        t2 = wall_time();

        // Verificacao do resultado
        if (Correto(h_tabulIn, tam))
            printf("**RESULTADO CORRETO**\n");
        else
            printf("**RESULTADO ERRADO**\n");

        t3 = wall_time();
        printf("tam=%d; tempos: init=%7.7f, comp=%7.7f, fim=%7.7f, tot=%7.7f \n",
               tam, t1 - t0, t2 - t1, t3 - t2, t3 - t0);

        // Limpeza de memoria
        free(h_tabulIn);
        free(h_tabulOut);
        CUDA_CHECK(hipFree(d_tabulIn));
        CUDA_CHECK(hipFree(d_tabulOut));
    }

    return 0;
}